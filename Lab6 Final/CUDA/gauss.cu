#include "hip/hip_runtime.h"
#include "GaussBaseFunc.h"

__global__ void division_kernel(float* mat, int k, int n) 
{
	float pivot = mat[k * n + k];
	//��i��ʼ��Ϊ�߳������������̸߳��СΪ��1��1�������Բ���Ҫ����gridDim�й���Ϣ
	for (int i = k + 1 + threadIdx.x; i  < n; i += blockDim.x)//ͨ��ѭ�����ֵķ�ʽ��������
		mat[k * n + i] /= pivot;
}

__global__ void eliminate_kernel(float* mat, int k, int n) 
{
	//��Ž��̸��𽫹�һ������δ����ĶԽ���Ԫ����Ϊ 1
	if (blockIdx.x == 0 && threadIdx.x == 0) mat[k * n + k] = 1;

	for (int i = k + 1 + blockIdx.x; i < n; i += gridDim.x){
		float pivot = mat[i * n + k];
		for (int j = k + 1 + threadIdx.x; j < n; j += blockDim.x) {
			mat[i * n + j] -= pivot * mat[k * n + j];
		}
		__syncthreads();//����ͬ��
		if (threadIdx.x == 0) mat[i * n + k] = 0;//����ֵ�·���Ӧλ����0
	}	
}

double cuda(int n, float* mat, int blockSize)
{
	size_t size = n * n * sizeof(float);

	//��ʼ��ʱ
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// ����GPU�ڴ沢�����ݴ������ڴ渴�Ƶ�GPU�ڴ�
	float* d_m;
	hipMalloc((void**)&d_m, n * n * sizeof(float));
	hipMemcpy(d_m, mat, size, hipMemcpyHostToDevice);

	for (int k = 0; k < n; k++) {
		division_kernel << <1, blockSize >> > (d_m, k, n);
		hipDeviceSynchronize();
		eliminate_kernel << <1, blockSize >> > (d_m, k, n);
		hipDeviceSynchronize();
	}

	// �����ݴ�GPU�ڴ渴�Ƶ������ڴ�
	hipMemcpy(mat, d_m, size, hipMemcpyDeviceToHost);

	//������ʱ
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	//�ͷ��ڴ�
	hipFree(d_m);

	return elapsedTime / 1000.0;
}

double cuda_plus(int n, float* mat, int blockSize) {
	size_t size = n * n * sizeof(float);

	//��ʼ��ʱ
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//�õ��ദ�����������Ӷ�ȷ���������ά��
	int deviceId;
	int numberOfSMs;
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	// ����GPU�ڴ沢�����ݴ������ڴ渴�Ƶ�GPU�ڴ�
	float* d_m;
	hipMalloc((void**)&d_m, n * n * sizeof(float));
	hipMemcpy(d_m, mat, size, hipMemcpyHostToDevice);

	//ִ�к˺���
	for (int k = 0; k < n; k++) {
		division_kernel << <1, blockSize >> > (d_m, k, n);
		hipDeviceSynchronize();
		eliminate_kernel << <numberOfSMs, blockSize >> > (d_m, k, n);
		hipDeviceSynchronize();
	}

	// �����ݴ�GPU�ڴ渴�Ƶ������ڴ�
	hipMemcpy(mat, d_m, size, hipMemcpyDeviceToHost);

	//������ʱ
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	//�ͷ��ڴ�
	hipFree(d_m);

	return elapsedTime / 1000.0;
}

int main()
{
	freopen("out.xls", "w", stdout);
	//timing(cuda, 2048, 1024);
	//timing(serial, 2048, 1024);

	string nameArr[] = { "cuda_plus" };
	double(*funcArr[])(int, float*, int) = {cuda_plus };
	//timingAllMatSize(nameArr, funcArr, 2, 2048, 1024);
	timingAllBlockSize(nameArr, funcArr, 1, 2048, 1024);
}
