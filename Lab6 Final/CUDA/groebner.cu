#include "hip/hip_runtime.h"
#include "EliminateClass.h"

int Nmat;//�������
int Nee;//����Ԫ������
int Ner;//��Ԫ������
int Ncol;//תΪuint���Ӧ������

Eliminater* globalEliminater;//ȫ����Ԫ����
Eliminatee* globalEliminatee;//ȫ�ֱ���Ԫ����
Eliminater* eliminater;//��Ԫ����
Eliminatee* eliminatee;//����Ԫ����

//��ȡ����
void dataInit(string DATA)
{
   //��ʼ����Ϣ
   stringstream ss(DATA);
   char c;
   ss >> Nmat >> c >> Nmat >> c >> Ner >> c >> Nee;
   if (DATA == "8_23045_18748_14325") Nmat = 23075;
   if (DATA == "9_37960_29304_14921") Nee = 14291;
   Ncol = (Nmat - (Nmat & 0xff) + 0x100) >> 5;

   eliminater = new Eliminater(Nmat);
   eliminatee = new Eliminatee(Nmat, Nee);
   globalEliminater = new Eliminater(DATAPATH + DATA + Para1, Nmat, Ner);
   globalEliminatee = new Eliminatee(DATAPATH + DATA + Para2, Nmat, Nee);
}

void timing(double(*func)(int), int dataIndex, int blockSize = 1024)
{
   string DATA = dataArr[dataIndex - 1];
   dataInit(DATA);

   double totalTime = 0.0f;
   //�Ƚ���WARMUP�������������ʱ��ʹ�ò���ʱ����Ӿ�ȷ
   for (int run = 0; run < REPEAT_NUM + WARMUP; run++) {
       eliminater->copy(*globalEliminater);
       eliminatee->copy(*globalEliminatee);
       float duration = func(blockSize);
       if (run >= WARMUP) totalTime += duration;
   }
   cout << totalTime / REPEAT_NUM << '\t';

   //�����
   eliminatee->check(DATAPATH + DATA + Para3);

   //�ͷ��ڴ�
   delete eliminatee;
   delete eliminater;
   delete globalEliminater;
   delete globalEliminatee;
}

void timingAll(string nameArr[], double(*funcArr[])(int), int funcNum, int start, int end, int blockSize = 1024)
{
   //�����ͷ
   cout << "Problem\t";
   for (int i = 0; i < funcNum; ++i)
       cout << nameArr[i] << '\t';
   cout << endl;

   for (int dataIndex = start; dataIndex <= end; ++dataIndex) {
       cout << dataIndex << '\t';
       string DATA = dataArr[dataIndex - 1];
       dataInit(DATA);

       for (int funcIndex = 0; funcIndex < funcNum; ++funcIndex) {
           double totalTime = 0.0f;
           //�Ƚ���WARMUP�������������ʱ��ʹ�ò���ʱ����Ӿ�ȷ
           for (int run = 0; run < REPEAT_NUM + WARMUP; run++) {
               eliminater->copy(*globalEliminater);
               eliminatee->copy(*globalEliminatee);
               float duration = funcArr[funcIndex](blockSize);
               if (run >= WARMUP) totalTime += duration;
           }
           cout << totalTime / REPEAT_NUM << '\t';
           //�����
           eliminatee->check(DATAPATH + DATA + Para3);
       }
       cout << endl;

       delete eliminatee;
       delete eliminater;
       delete globalEliminater;
       delete globalEliminatee;
   }
}

double serial1(int blockSize = 0)
{
   auto start = chrono::high_resolution_clock::now();

   uint* elier, * eliee;
   for (int i = 0; i < Nee; ++i)
   {
       eliee = eliminatee->m_elieeList[i];
       for (int j = Nmat - 1; j >= 0; --j)
       {
           if (!getBit(eliee, j)) continue;
           if (eliminater->m_isElierList[j])
           {
               elier = eliminater->m_elierList[j];
               for (int k = 0; k < Ncol; ++k) eliee[k] ^= elier[k];
           }
           else
           {
               eliminater->m_elierList[j] = eliee;
               eliminater->m_isElierList[j] = true;
               eliminatee->m_isElierList[i] = true;
               break;
           }
       }
   }

   auto stop = chrono::high_resolution_clock::now();
   auto duration = chrono::duration_cast<chrono::microseconds>(stop - start) / 1000000.0;
   return duration.count();
}

double serial2(int blockSize = 0)
{
   auto start = chrono::high_resolution_clock::now();

   uint* elier, * eliee;
   for (int j = Nmat - 1; j >= 0; --j) { // ������Ԫ��
       elier = eliminater->m_elierList[j];
       if (!eliminater->m_isElierList[j]) { // ��������ڶ�Ӧ��Ԫ���򽫱���Ԫ������
           int i;
           for (i = 0; i < Nee; ++i)
           {
               if (eliminatee->m_isElierList[i])
                   continue;
               uint* eliee = eliminatee->m_elieeList[i];
               if (getBit(eliee, j))
               {
                   eliminater->m_elierList[j] = eliee;
                   eliminater->m_isElierList[j] = true;
                   eliminatee->m_isElierList[i] = true;
                   elier = eliee;
                   break;
               }
           }
           if (i == Nee) continue;
       }
       for (int i = 0; i < Nee; ++i)
       { // ��������Ԫ��
           if (eliminatee->m_isElierList[i])
               continue;
           eliee = eliminatee->m_elieeList[i];
           if (getBit(eliee, j))
           { // �����ǰ����Ҫ��Ԫ
               for (int k = 0; k < Ncol; ++k)
                   eliee[k] ^= elier[k];
           }
       }
   }

   auto stop = chrono::high_resolution_clock::now();
   auto duration = chrono::duration_cast<chrono::microseconds>(stop - start) / 1000000.0;
   return duration.count();
}


double serial3(int blockSize = 0)
{
   auto start = chrono::high_resolution_clock::now();

   int endj = Nmat - 1;
   uint* elier, * eliee;
   for (int j = endj; j >= 0; j = endj) { // ������Ԫ��

       //��groupSizeΪ�������ҵ�����ֱ��������Ԫ������
       for (int i = 0; i < groupSize; ++i, --endj)
           if (endj < 0 || !eliminater->m_isElierList[endj])
               break;

       if (j == endj) {// ��������ڶ�Ӧ��Ԫ���򽫱���Ԫ������
           int i;
           for (i = 0; i < Nee; ++i)
           {
               if (eliminatee->m_isElierList[i])
                   continue;
               uint* eliee = eliminatee->m_elieeList[i];
               if (getBit(eliee, j))
               {
                   eliminater->m_elierList[j] = eliee;
                   eliminater->m_isElierList[j] = true;
                   eliminatee->m_isElierList[i] = true;
                   break;
               }
           }
           if (i == Nee) --endj;
           continue;
       }

       for (int jj = j; jj > endj; --jj) {
           elier = eliminater->m_elierList[jj];
           for (int i = 0; i < Nee; ++i) { // ��������Ԫ��
               if (eliminatee->m_isElierList[i])
                   continue;
               eliee = eliminatee->m_elieeList[i];
               if (getBit(eliee, jj))
               { // �����ǰ����Ҫ��Ԫ
                   for (int k = 0; k < Ncol; ++k)
                       eliee[k] ^= elier[k];
               }
           }
       }
   }

   auto stop = chrono::high_resolution_clock::now();
   auto duration = chrono::duration_cast<chrono::microseconds>(stop - start) / 1000000.0;
   return duration.count();
}

__global__ void eliminate_kernel(uint* elieeData, uint* eliers, bool* isElierList, int j, int endj, int Nee, int Ncol)
{
   uint* elier, * eliee;
   for (int jj = j; jj > endj; --jj) {
       elier = eliers + (jj - endj - 1) * Ncol;
       for (int i = blockIdx.x; i < Nee; i += gridDim.x) {
           if (isElierList[i])
               continue;
           eliee = elieeData + i * Ncol;
           if (eliee[jj >> 5] & (0x80000000 >> (jj & 0x1f)))
           { // �����ǰ����Ҫ��Ԫ
               for (int k = threadIdx.x; k < Ncol; k += blockDim.x)//ͨ��ѭ�����ֻ�������
                   eliee[k] ^= elier[k];
           }
       }
       __syncthreads();//����ͬ��
   }
}

double cuda(int blockSize) {
   //��ʼ��ʱ
   float elapsedTime;
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   //�õ��ദ�����������Ӷ�ȷ���������ά��
   int deviceId;
   int numberOfSMs;
   hipGetDevice(&deviceId);
   hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

   // ����GPU�ڴ�
   bool* isElierList;
   uint* elieeData, * eliers;
   hipMalloc((void**)&isElierList, Nee);
   hipMalloc((void**)&elieeData, Nee * Ncol * sizeof(uint));
   hipMalloc((void**)&eliers, groupSize * Ncol * sizeof(uint));

   //�����ݴ������ڴ渴�Ƶ�GPU�ڴ�
   hipMemcpy(elieeData, eliminatee->dataVector, Nee * Ncol * sizeof(uint), hipMemcpyHostToDevice);

   //ִ�к˺���
   int endj = Nmat - 1;
   for (int j = endj; j >= 0; j = endj) { // ������Ԫ��

       //��groupSizeΪ�������ҵ�����ֱ��������Ԫ������
       for (int i = 0; i < groupSize; ++i, --endj)
           if (endj < 0 || !eliminater->m_isElierList[endj])
               break;

       if (j == endj) {// ��������ڶ�Ӧ��Ԫ���򽫱���Ԫ������
           int i;
           for (i = 0; i < Nee; ++i)
           {
               if (eliminatee->m_isElierList[i])
                   continue;
               uint* eliee = eliminatee->m_elieeList[i];
               if (getBit(eliee, j))
               {
                   memcpy(eliminater->m_elierList[j], eliee, Ncol * sizeof(uint));
                   eliminater->m_isElierList[j] = true;
                   eliminatee->m_isElierList[i] = true;
                   break;
               }
           }
           if (i == Nee) --endj;
           continue;
       }

       //����Ԫ�Ӻ��Ƿ��������Ϣ���ݵ�GPU��
       hipMemcpy(eliers, eliminater->dataVector + (endj + 1) * Ncol, (j - endj) * Ncol * sizeof(uint), hipMemcpyHostToDevice);
       hipMemcpy(isElierList, eliminatee->m_isElierList, Ncol , hipMemcpyHostToDevice);

       //������Ԫ
       eliminate_kernel << <numberOfSMs, blockSize >> > (elieeData, eliers, isElierList, j, endj, Nee, Ncol);

       //�ȴ����н�����Ԫ���
       hipDeviceSynchronize();


       if (!eliminater->m_isElierList[endj]) {//�ж��Ƿ񽫱���Ԫ�и��ƻ�CPU��
           hipMemcpy(eliminatee->dataVector, elieeData, Nee * Ncol * sizeof(uint), hipMemcpyDeviceToHost);
       }
   }


   // �����ݴ�GPU�ڴ渴�Ƶ������ڴ�
   hipMemcpy(eliminatee->dataVector, elieeData, Nee * Ncol * sizeof(uint), hipMemcpyDeviceToHost);

   //������ʱ
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start, stop);

   //�ͷ��ڴ�
   hipFree(isElierList);
   hipFree(elieeData);
   hipFree(eliers);

   return elapsedTime / 1000.0;
}

int main()
{
   //timing(serial1, 2, 128);

   string namearr[] = { "serial1","serial3","cuda"};
   double(*funcArr[])(int) = { serial1,serial3,cuda };
   timingAll(namearr, funcArr, 3, 11, 11);
}